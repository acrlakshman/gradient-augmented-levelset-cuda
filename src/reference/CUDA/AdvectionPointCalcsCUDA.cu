#include "hip/hip_runtime.h"
/*
This file is an implementation of all the functions that work on a single node
in the 2D grid and update the level set.
*/

#include "HermiteCUDA.cu"
#include "TimeSteppingMethodsCUDA.cu"

#ifndef _AdvectionPointCalcsCUDA_h
#define _AdvectionPointCalcsCIDA_h

__global__ void advection_point_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int nx,
        unsigned int t, double dt, double T_period, unsigned int TileSize)
{
    double c1 = (1/6.0);
    double c2 = (1/6.0);
    double c3 = (2/3.0);    //RK-3 constants
    
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    
    double ux = Velx(x[index_x], y[index_y], (t + 1) * dt, T_period);
    double vy = Vely(x[index_x], y[index_y], (t + 1) * dt, T_period);
    
    unsigned int indexToWrite = index_y * nx + index_x;
    
    // Advected Points - 1 Step
    xadv[indexToWrite] = x[index_x] - ux * dt;
    yadv[indexToWrite] = y[index_y] - vy * dt;
    
    double ux1 = Velx(xadv[indexToWrite], yadv[indexToWrite], t * dt, T_period);
    double vy1 = Vely(xadv[indexToWrite], yadv[indexToWrite], t * dt, T_period);
    
    // 2 Step
    xadv[indexToWrite] = x[index_x] - ((dt)/2.0 * (0.5 * ux + 0.5 * ux1));
    yadv[indexToWrite] = y[index_y] - ((dt)/2.0 * (0.5 * vy + 0.5 * vy1));
    
    double ux2 = Velx(xadv[indexToWrite], yadv[indexToWrite],(t * dt + dt/2.0), T_period);
    double vy2 = Vely(xadv[indexToWrite], yadv[indexToWrite],(t * dt + dt/2.0), T_period);
    
    // 3 Step
    xadv[indexToWrite] = x[index_x] - (dt * (c1 * ux + c2 * ux1 + c3 * ux2));
    yadv[indexToWrite] = y[index_y] - (dt * (c1 * vy + c2 * vy1 + c3 * vy2));
}

__device__ unsigned int locationAlgo(double *x, double xadv, unsigned int nx)
{
    unsigned int location = 0;
    while (x[location] < xadv && location < nx)
        location++;
    if(location == 0)
        return location;
    else
        return location-1;
}

__global__ void find_advection_point_location_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int nx, unsigned int ny,
        unsigned int *cellx, unsigned int *celly, unsigned int *tracker, double xlim1, double xlim2, double ylim1,
        double ylim2, unsigned int TileSize)
{
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    
    unsigned int indexToWrite = index_y * nx + index_x;
    
    bool xoutofbounds = false;
    bool youtofbounds = false;
    
    if(!((xadv[indexToWrite] > xlim1) && (xadv[indexToWrite] < xlim2)))
        xoutofbounds = true;
    if(!((yadv[indexToWrite] > ylim1) && (yadv[indexToWrite] < ylim2)))
        youtofbounds = true;
    
    if(!xoutofbounds && !youtofbounds)
    {
        tracker[indexToWrite] = 1;
        cellx[indexToWrite] = locationAlgo(x,xadv[indexToWrite],nx);
        celly[indexToWrite] = locationAlgo(y,yadv[indexToWrite],ny);
    }
    else
        if(!xoutofbounds && youtofbounds)
        {
            tracker[indexToWrite] = 2;
            cellx[indexToWrite] = locationAlgo(x,xadv[indexToWrite],nx);
            if(yadv[indexToWrite] <= ylim1)
                celly[indexToWrite] = 0;
            else
                if(yadv[indexToWrite] >= ylim2)
                    celly[indexToWrite] = ny-2;
        }
        else
            if(xoutofbounds && !youtofbounds)
            {
                tracker[indexToWrite] = 3;
                celly[indexToWrite] = locationAlgo(y,yadv[indexToWrite],ny);
                if(xadv[indexToWrite] <= xlim1)
                    cellx[indexToWrite] = 0;
                else
                    if(xadv[indexToWrite] >= xlim2)
                        cellx[indexToWrite] = nx-2;
            }
            else
                if(xoutofbounds && youtofbounds)
                    tracker[indexToWrite] = 4;
}

__global__ void update_levelset_data_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int *cellx,
        unsigned int *celly, unsigned int *tracker, unsigned int t, double dt, double *tempphi, double *temppsix,
        double *temppsiy, double *temppsixy, double *mphi, double *mpsix, double *mpsiy, char psischeme[],
        char backtrace_scheme[], double T_period, unsigned int nx, unsigned int ny, unsigned int TileSize)
{
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    
    unsigned int indexToWrite = index_y * nx + index_x;
    
    double dx = x[2] - x[1];
    double dy = y[2] - y[1];
    
    double phi[4], psix[4], psiy[4], psixy[4];
    
    unsigned int cellindex_x = cellx[indexToWrite];
    unsigned int cellindex_y = celly[indexToWrite];
    
    unsigned int cellindex = cellindex_x + cellindex_y * nx;
    
    // Storing the four values for four nodes of each cell
    phi[0] = mphi[cellindex];            psix[0] = mpsix[cellindex];          psiy[0] = mpsiy[cellindex];          psixy[0] = temppsixy[cellindex];
    phi[1] = mphi[cellindex + 1];          psix[1] = mpsix[cellindex + 1];        psiy[1] = mpsiy[cellindex + 1];        psixy[1] = temppsixy[cellindex + 1];
    phi[2] = mphi[cellindex + nx];          psix[2] = mpsix[cellindex + nx];        psiy[2] = mpsiy[cellindex + nx];        psixy[2] = temppsixy[cellindex + nx];
    phi[3] = mphi[cellindex + nx + 1];        psix[3] = mpsix[cellindex + nx + 1];      psiy[3] = mpsiy[cellindex + nx + 1];      psixy[3] = temppsixy[cellindex + nx + 1];
    // Node value assignment ends
    
    // Storing the coordinates of first node of the working cell
    double xo = x[cellindex_x], yo = y[cellindex_y];
    
    switch(tracker[indexToWrite]){
        case 1:{
            tempphi[indexToWrite] = hp(phi, psix, psiy, psixy, xadv[indexToWrite], yadv[indexToWrite], xo, yo, dx, dy);
            double rootpsix = hermx(phi,psix,psiy,psixy,xadv[indexToWrite],yadv[indexToWrite],xo,yo,dx, dy);
            double rootpsiy = hermy(phi,psix,psiy,psixy,xadv[indexToWrite],yadv[indexToWrite],xo,yo,dx, dy);
            
            //Commenting out the options of psi scheme
            //SuperConsistent is being used now irrespective of specification given in
            //GALS_Advection.cu
            //The if/else functionality may be added later
            //if(strcmp("Heuns",psischeme) == 0)
            //Heuns_internal(x[index_x],y[index_y],xadv[indexToWrite],yadv[indexToWrite],rootpsix,rootpsiy,t,dt,T_period,temppsix,temppsiy,indexToWrite);
            //else if(strcmp("SuperConsistent",psischeme) == 0)
            SuperConsistentScheme(x[index_x],y[index_y],rootpsix,rootpsiy,t,dt,T_period,backtrace_scheme,temppsix,temppsiy,indexToWrite);
            
            break;
        } // end of case 1
        
        case 2:{
            double rootpsix = hermx(phi,psix,psiy,psixy,xadv[indexToWrite], y[index_y],xo,yo,dx, dy);
            double rootpsiy = temppsiy[indexToWrite];
            tempphi[indexToWrite] = hp(phi, psix, psiy, psixy, xadv[indexToWrite], y[index_y], xo, yo, dx, dy) - dt * Vely(x[index_x], y[index_y], t * dt, T_period) * rootpsiy;
            temppsix[indexToWrite] = Heuns_X(x[index_x],y[index_y],rootpsix,rootpsiy,temppsixy[indexToWrite],temppsix[indexToWrite],t,dt,T_period);
            break;
        }   // end of case 2
        
        case 3:{
            double rootpsix = temppsix[indexToWrite];
            double rootpsiy = hermy(phi,psix,psiy,psixy,x[index_x],yadv[indexToWrite],xo,yo,dx, dy);
            tempphi[indexToWrite] = hp(phi, psix, psiy, psixy, x[index_x],yadv[indexToWrite], xo, yo, dx, dy) - dt * Velx(x[index_x], y[index_y], t * dt, T_period) * rootpsix;
            temppsiy[indexToWrite] = Heuns_Y(x[index_x],y[index_y],rootpsix,rootpsiy,temppsixy[indexToWrite],temppsiy[indexToWrite],t,dt,T_period);
            break;
        }   // end of case 3
        
        case 4:{
            tempphi[indexToWrite] = tempphi[indexToWrite] - dt * (Velx(x[index_x], y[index_y], t * dt, T_period) * temppsix[indexToWrite] + Vely(x[index_x], y[index_y], t * dt, T_period) * temppsiy[indexToWrite]);
            break;
        } //end of case4
        
        default:{break;}
    }   // end of switch loop
}

__global__ void update_mixed_derivatives(double *temppsix, double *temppsiy, double *temppsixy,
        unsigned int nx, unsigned int ny, double dx, double dy, unsigned int TileSize)
{
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    
    unsigned int indexToWrite = index_y * nx + index_x;
    
    if ((index_y == 0 || index_y == ny - 1) && (index_x != 0 && index_x != nx - 1))
        temppsixy[indexToWrite] = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1])/(2 * dx);
    else
        if ((index_y != 0 && index_y != ny - 1) && (index_x == 0 || index_x == nx - 1))
            temppsixy[indexToWrite] = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx])/(2 * dy);
        else
            if((index_y == 0 || index_y == ny - 1) && (index_x == 0 || index_x == nx - 1)){
                if(index_y == 0 && index_x == 0){
                    double d1 = (temppsiy[1] - temppsiy[0])/dx;
                    double d2 = (temppsix[nx] - temppsix[0])/dy;
                    double d3 = (temppsix[nx+1] - temppsix[1])/dy;
                    double d4 = (temppsiy[nx+1] - temppsiy[nx])/dx;
                    temppsixy[indexToWrite] = 0.75 * (d1 + d2) - 0.25 * (d3 + d4);
                }
                else if(index_y == 0 && index_x == nx-1){
                    double d1 = (temppsiy[nx-1] - temppsiy[nx-2])/dx;
                    double d2 = (temppsix[nx+nx-2] - temppsix[nx-2])/dy;
                    double d3 = (temppsix[nx+nx-1] - temppsix[nx-1])/dy;
                    double d4 = (temppsiy[nx+nx-1] - temppsiy[nx+nx-2])/dx;
                    temppsixy[indexToWrite] = 0.75 * (d1 + d3) - 0.25 * (d2 + d4);
                    
                }
                else if(index_y == ny-1 && index_x == 0){
                    double d1 = (temppsiy[nx *(ny-2) + 1] - temppsiy[nx *(ny-2)])/dx;
                    double d2 = (temppsix[nx *(ny-1)] - temppsix[nx *(ny-2)])/dy;
                    double d3 = (temppsix[nx *(ny-1)] - temppsix[nx *(ny-2) + 1])/dy;
                    double d4 = (temppsiy[nx *(ny-1) + 1] - temppsiy[nx *(ny-1)])/dx;
                    temppsixy[indexToWrite] = 0.75 * (d2 + d4) - 0.25 * (d3 + d1);
                    
                }
                else if(index_y == ny-1 && index_x == nx-1){
                    double d1 = (temppsiy[nx *(ny-2) + nx - 1] - temppsiy[nx *(ny-2) + nx - 2])/dx;
                    double d2 = (temppsix[nx *(ny-1) + nx - 2] - temppsix[nx *(ny-2) + nx - 2])/dy;
                    double d3 = (temppsix[nx *(ny-1) + nx - 1] - temppsix[nx *(ny-2) + nx - 1])/dy;
                    double d4 = (temppsiy[nx *(ny-1) + nx - 1] - temppsiy[nx *(ny-1) + nx - 2])/dx;
                    temppsixy[indexToWrite] = 0.75 * (d3 + d4) - 0.25 * (d1 + d2);
                }
            }
            else{
                double dxy1 = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1])/(2 * dx);
                double dxy2 = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx])/(2 * dy);
                temppsixy[indexToWrite] = (dxy1 + dxy2)/2.0;
            }
    
}
#endif
