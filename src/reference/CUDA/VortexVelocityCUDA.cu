#include "hip/hip_runtime.h"
/*
This file defines the functions required to calculate the analytical velocity at
any point in space and time for the given 2D domain
This file is for a Vortex type velocity field. For any other field, this file needs
to be modified completely.
*/

#ifndef _VortexVelocityCUDA_cu
#define _VortexVelocityCUDA_cu

__device__ __host__ double Velx(double x, double y, double t, double T_period){
    double temp;
    temp = pow(sin(pi * x),2) * sin(2 * pi * y) * cos(pi * t/T_period);
    return temp;
}

__device__ __host__ double Vely(double x, double y,double t, double T_period){
    double temp;
    temp = -pow(sin(pi * y),2) * sin(2 * pi * x) * cos(pi * t/T_period);
    return temp;
}

__device__ double gradUx(double x, double y, double t, double T_period){
    double temp;
    temp = pi * sin(2 * pi * y) * sin(2 * pi * x) * cos(pi * t/T_period);
    return temp;
}

__device__ double gradUy(double x, double y, double t, double T_period){
    double temp;
    temp = 2 * pi * pow(sin(pi * x),2) * cos(2 * pi * y) * cos(pi * t/T_period);
    return temp;
}

__device__ double gradVx(double x, double y, double t, double T_period){
    double temp;
    temp = -2 * pi * pow(sin(pi * y),2) * cos(2 * pi * x) * cos(pi * t/T_period);
    return temp;
}

__device__ double gradVy(double x, double y, double t, double T_period){
    double temp;
    temp = -pi * sin(2 * pi * y) * sin(2 * pi * x) * cos(pi * t/T_period);
    return temp;
}
#endif
