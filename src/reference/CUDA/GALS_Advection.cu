#include "hip/hip_runtime.h"
//  GALS - Before compiling the program, update the section of this program that is in the beginning of main and update all the initializer files
//
//  Created by Raunak Bardia, Chia-Wei Kuo, and Arpit Agarwal on December 20, 2017.
//
// Implementing GALS for a given initial level set function
// in a specified velocity field for a grid of cells
//
// Given -
// 1. Defining function at t = 0 which implies that phi and psi values are available for all node points at t = 0
// 2. Given velocity for the complete domain at all times
//
// All required data is stored in separate 2D matrices of phi, psix, psiy and psixy
// Boundary Condition grad(velocity).n > 0

/* * * * * * * * * * * * * *  CUDA IMPLEMENTATION * * * * * * * * * * * * * * */
// THIS IMPLEMENTATION WON'T WORK IF THE GRID IS SMALLER THAN (2 X 2)
#include <iostream>
#include <iomanip>
#include <math.h>
#include <stdio.h>
#include <fstream>
#include <sys/time.h>
#include <time.h>
#include <string.h>
#include <vector>
#include <tuple>
#include <hip/hip_runtime.h>
#include <chrono>
#include "Allocation.h"

//Including Kernel
#include "AdvectionPointCalcsCUDA.cu"
#include "VortexVelocityCUDA.cu"

// y direction is the first index of array, x direction is the second index of array

using namespace std;

int main(){
    
    /* UPDATE ALL THE FOLLOWING VALUES */
    double xlim1 = 0.0;                       //Lower limit on x-axis
    double xlim2 = 1.0;                      //Upper limit on x-axis
    unsigned int nx = 128;                         //Number of nodes in x-direction INCLUDING THE EXTREME VALUES
    
    double ylim1 = 0.0;                       //Lower limit on y-axis
    double ylim2 = 1.0;                     //Upper limit on y-axis
    unsigned int ny = 128;                        //Number of nodes INCLUDING THE EXTREME VALUES
    
    double dt = 0.5 * 1.0/128.0;                     //Length of time step
    double Tfinal = 1.0;                    //Total time period for the simulation
    unsigned int option = 1;                         //Option - if you need animation initialize at 1 else initialize at 2
    unsigned int printstep = 16;                      //How frequently do you want to store the images (every nth time step)
    char psischeme[] = "SuperConsistent";   //'SuperConsistent' or 'Heuns'
    char backtrace_scheme[] = "RK3" ;      //'Euler' or 'RK3'
    double T_period = 1.0;                  //Period of the velocity field
    unsigned int TileSize = 16;
    
    //---------------------------------------------------------------------------------------------------------
    //MAKE SURE THAT YOU HAVE ENOUGH MEMORY SPACE IF YOU ARE STORING A LOT OF TIME STEP VALUES BECAUSE IT STORES ACROSS GRID POINTS FOR EACH PRINTSTEP
    
    /* USER UPDATE OVER */
    unsigned long gridmemory = nx * ny * sizeof(double);
    unsigned long gridmemoryint = nx * ny * sizeof(unsigned int);
    unsigned int n = Tfinal/dt; //Number of time steps
    if(option != 1)
        printstep = n;
    
    dim3 dimBlock(TileSize, TileSize);
    dim3 dimGrid(nx/dimBlock.x, ny/dimBlock.y);
    
    // Node Locations
    double dx = (xlim2 - xlim1)/(nx - 1);
    double dy = (ylim2 - ylim1)/(ny - 1);
    double* x = (double*) malloc(nx * sizeof(double));
    double* y = (double*) malloc(ny * sizeof(double));
    gridnodes(x,y,xlim1,ylim1,dx,dy,nx,ny);
    double *devicex, *devicey;
    // allocate device memory for x and y
    hipMalloc((void**)&devicex,nx * sizeof(double));	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&devicey,ny * sizeof(double));	// Allocating GPU memory for the y-node values
    // Copy data from host to GPU
    hipMemcpy(devicex, x, nx * sizeof(double), hipMemcpyHostToDevice);	// Writing to device memory
    hipMemcpy(devicey, y, ny * sizeof(double), hipMemcpyHostToDevice);	// Writing to device memory
    
    // level set matrices
    double* mphi = (double*) malloc(gridmemory);
    double* mpsix = (double*) malloc(gridmemory);
    double* mpsiy = (double*) malloc(gridmemory);
    double* mpsixy = (double*) malloc(gridmemory);
    
    double *masterdphi, *masterdpsix, *masterdpsiy, *masterdpsixy;
    
    // allocate device memory for integer grids
    hipMalloc((void**)&masterdphi,gridmemory);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&masterdpsix,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&masterdpsiy,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&masterdpsixy,gridmemory);	// Allocating GPU memory for the y-node values
    double *dphi, *dpsix, *dpsiy, *dpsixy;
    // allocate device memory for integer grids
    hipMalloc((void**)&dphi,gridmemory);	// Allocating GPU memory for the x-node values
    hipMalloc((void**)&dpsix,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&dpsiy,gridmemory);	// Allocating GPU memory for the y-node values
    hipMalloc((void**)&dpsixy,gridmemory);	// Allocating GPU memory for the y-node values
    
    // Initializing at t = 0
    allocate_levelset_matrices_CUDA<<<dimGrid, dimBlock>>>(masterdphi, masterdpsix, masterdpsiy, masterdpsixy, devicex, devicey, nx, ny); //Initializing level set matrices
    allocate_levelset_matrices_CUDA<<<dimGrid, dimBlock>>>(dphi, dpsix, dpsiy, dpsixy, devicex, devicey, nx, ny); //Initializing level set matrices
    // Initializing at t = 0
    allocate_levelset_matrices(mphi,mpsix,mpsiy,mpsixy,x,y,nx,ny); //Initializing level set matrices
    
    // Removing existing files with these names if any
    remove("phi.txt");
    remove("psix.txt");
    remove("psiy.txt");
    remove("psixy.txt");
    remove("details.txt");
    remove("Velocity_x.txt");
    remove("Velocity_y.txt");
    fileprint(mphi,mpsix,mpsiy,mpsixy,nx,ny,x,y,0.0,T_period);
    ofstream details;
    details.open("details.txt", ios::out | ios::app);
    details<< nx << "," << ny << "," << std::fixed << std::setprecision(10) << dx << "," << dy << "," << xlim1 << "," << xlim2 << "," << ylim1 << "," << ylim2 << "," << n << "," << dt << "," << printstep;
    details.close();
    
    ///*
    // TIME STEPPING LOOP
    // If only the initial and final profiles are needed
// This section will be deleted after a proper profiler is installed on my computer - Raunak Bardia
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float t_calcpts=0.0,t_findpts=0.0,t_update=0.0,t_mixed=0.0,t_copy=0.0,t_transfer=0.0,tempt=0.0;
    auto tbegin = chrono::high_resolution_clock::now();
    
//
    for(unsigned int t = 0; t < n; t++){
        
        double *dxadv, *dyadv;
        // allocate device memory for x and y
        hipMalloc((void**)&dxadv,gridmemory);	// Allocating GPU memory for the x-node values
        hipMalloc((void**)&dyadv,gridmemory);	// Allocating GPU memory for the y-node values
        
        unsigned int *dcellx, *dcelly,*dtracker;
        // allocate device memory for integer grids
        hipMalloc((void**)&dcellx,gridmemoryint);	// Allocating GPU memory for the x-node values
        hipMalloc((void**)&dcelly,gridmemoryint);	// Allocating GPU memory for the y-node values
        hipMalloc((void**)&dtracker,gridmemoryint);	// Allocating GPU memory for the y-node values
        
        // Find the point from which advection occurs at this time step
        hipEventRecord(startEvent,0);
        advection_point_cuda<<<dimGrid,dimBlock>>>(devicex,devicey,dxadv,dyadv,nx,t,dt,T_period,TileSize);
        hipEventRecord(stopEvent,0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&tempt, startEvent, stopEvent);
        t_calcpts += tempt;
        
        // Find the cell in which those advection points lie
        hipEventRecord(startEvent,0);
        find_advection_point_location_cuda<<<dimGrid,dimBlock>>>(devicex,devicey,dxadv,dyadv,nx,ny,dcellx,dcelly,dtracker,xlim1,xlim2,ylim1,ylim2,TileSize);
        hipEventRecord(stopEvent,0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&tempt, startEvent, stopEvent);
        t_findpts += tempt;
        
        // Update the level set values
        hipEventRecord(startEvent,0);
        update_levelset_data_cuda<<<dimGrid,dimBlock>>>(devicex, devicey, dxadv, dyadv, dcellx, dcelly, dtracker, t, dt, dphi, dpsix, dpsiy, dpsixy, masterdphi, masterdpsix, masterdpsiy,psischeme,backtrace_scheme,T_period,nx,ny,TileSize);
        hipEventRecord(stopEvent,0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&tempt, startEvent, stopEvent);
        t_update += tempt;
        
        // Create another copy to preserve data which gets modified on the fly in next loop
        hipEventRecord(startEvent,0);
        devicetodevicecopy<<<dimGrid,dimBlock>>>(dphi,dpsix,dpsiy,masterdphi,masterdpsix,masterdpsiy,nx,TileSize);
        hipEventRecord(stopEvent,0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&tempt, startEvent, stopEvent);
        t_copy += tempt;
        
        // Update the mixed derivatives now for the remaining grid points
        hipEventRecord(startEvent,0);
        update_mixed_derivatives<<<dimGrid,dimBlock>>>(dpsix, dpsiy, dpsixy, nx, ny, dx, dy,TileSize);
        hipEventRecord(stopEvent,0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&tempt, startEvent, stopEvent);
        t_mixed += tempt;
        
        hipDeviceSynchronize();
        
        //---------------------------------------------------------------------------------------------------------
        // Feeding phi, psix, psiy and psixy values in their respective files
        if((t+1) % printstep == 0)
        {
            hipEventRecord(startEvent,0);
            hipMemcpy(mphi, masterdphi, gridmemory, hipMemcpyDeviceToHost);       // Writing back to host memory
            hipMemcpy(mpsix, masterdpsix, gridmemory, hipMemcpyDeviceToHost);       // Writing back to host memory
            hipMemcpy(mpsiy, masterdpsiy, gridmemoryint, hipMemcpyDeviceToHost);      // Writing back to host memory
            hipEventRecord(stopEvent,0);
            hipEventSynchronize(stopEvent);
            hipEventElapsedTime(&tempt, startEvent, stopEvent);
            t_transfer += tempt;
            //hipMemcpy(mpsixy, masterdpsixy, gridmemoryint, hipMemcpyDeviceToHost);  // Writing back to host memory
            fileprint(mphi,mpsix,mpsiy,mpsixy,nx,ny,x,y,(t+1)*dt,T_period);
        }
        cout<< t+1;
        cout<< " Time Step Completed" <<'\n';
        
        //---------------------------------------------------------------------------------------------------------
        hipFree(dxadv);
        hipFree(dyadv);
        hipFree(dcellx);
        hipFree(dcelly);
        hipFree(dtracker);
    }  // end of time marching loop
    //*/
    auto tend = chrono::high_resolution_clock::now();
    float duration = chrono::duration_cast<chrono::nanoseconds>(tend-tbegin).count();
    duration = duration * pow(10.0,-6);
    cout << "Time taken for calculation of advection points = " << t_calcpts << '\n';
    cout << "Time taken for finding locatio of advection points in the grid = " << t_findpts << '\n';
    cout << "Time taken for hermite Update = " << t_update << '\n';
    cout << "Time taken for copying the matrix = " << t_copy << '\n';
    cout << "Time taken for calculation of mixed derivatives = " << t_mixed << '\n';
    cout << "Time taken for transfer of level set data = " << t_transfer << '\n';
    cout << "Total Duration of the Time Loop = " << duration << endl;
    
    return 0;
}
